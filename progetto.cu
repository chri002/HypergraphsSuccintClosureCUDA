#include "hip/hip_runtime.h"
/*
#########################################################################################
#																						#
#	    	  			    hypergraph transitive closure 								#
#	nvcc -rdc=true -lineinfo -Xcompiler -openmp .\progetto.cu -o progetto.exe			#
#			 			-D HIDE  : hide the output										#
#			 			-D DEBUG : show information on runtime							#
#			 			-D FILE_OUT : export graph to file								#
#			 			-D MAX_THREADS : max cuda threads 								#
#			 			-D MAX_BLOCKS : max cuda blocks 								#
#			 			-D TIME : enable time control	 								#
#			 			-D NO_INPUT : remove enter clic 								#
#																						#
#########################################################################################
*/

/*
Calcolo della chiusura transitiva succinta dato un ipergrafo H, in maniera parallela ove
	se ne disponga le risorse.
	Parallelismo CPU: per ogni vertice del ipergrafo si invoca BFS; merge dei vettori 
		ottenuti dalla BFS.
	Parallelismo GPU: la BFS per trovare tutti i nipoti del nodo.
	
Calculation of the succinct transitive closure given a hypergraph H, in a parallel way where
	there is the Hardware.
	CPU parallelism: BFS is invoked for each vertex of the hypergraph; merge of vectors
		obtained from the BFS.
	GPU parallelism: the BFS to find all the grandchildren of the node.

Work:
	progetto.exe "grafo.txt"
	
Experimental:
	inside copyaa remove comment around std::copy and compile with -std c++17 
	end comment the openmp for cycle
*/


#include <hip/hip_runtime.h>
#include <omp.h>
#include <vector>
#include <string>
#include <cstdio>
#include <cstdlib>
#include <istream>
#include <fstream>
#include <algorithm>
#include <chrono>
#include <execution>
#include <map>
#include <utility>

//Massimo numero di threads su GPU e Block su GPU
//numero threads su CPU


#ifndef MAX_THREADS
#define MAX_THREADS 128
#endif
#ifndef MAX_BLOCKS
#define MAX_BLOCKS 1
#endif
#ifdef NTHR
	int nThr = NTHR;
#else
	int nThr;
#endif

#ifdef TIME
std::chrono::high_resolution_clock::time_point begin;
std::chrono::high_resolution_clock::time_point end;
unsigned long durata,durataRead;

#endif


//gestione e cattura errori GPU
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


//Struttura iperarco

typedef struct{
	int* from;					//Insieme nodi provenienza CPU
	int* from_dev;				//Insieme nodi provenienza GPU
	int len_fr;					//lunghezza dei vettori precedenti
	int to;						//Nodo di arrivo dell'iperarco
} Hyperarc;

typedef struct{
	int* vectors;
	int len;
} Hypervector;

//Hyperarch comparison function for the unique function (NotEqual)
bool ne_compareTwoHyerarch(Hyperarc a, Hyperarc b)
{
	if(a.to!=b.to)
	return false;
	bool ok=true;
	int i;
	for(i=0; i<min(a.len_fr,b.len_fr) && ok; i++) {
		ok=ok && a.from[i]==b.from[i];
	}
	
	if(ok) return (a.len_fr==b.len_fr);
	else return false;
}
 
//Hyperarch comparison function for the unique function (EqualLess
bool compareTwoHyerarch(Hyperarc a, Hyperarc b)
{
	if(a.to!=b.to)
	return a.to < b.to;
	bool ok=true;
	int i,j=0;
	for(i=0; i<min(a.len_fr,b.len_fr) && ok; i++) {
		ok=ok && a.from[i]==b.from[i];
		j=i;
	}
	
	if(ok) return (a.len_fr<=b.len_fr);
	else return a.from[j]<b.from[j];
}

void printa(Hyperarc* hs, int l){
	printf("test %d\n",l);
	for(int i=0; i<l && hs[i].to!=-1; i++){
		printf("(HA {");
		for(int j=0; j<(hs)[i].len_fr; j++){
			printf("%d",(hs)[i].from[j]);
			if(j!=((hs)[i].len_fr)-1) printf(",");
		}
		printf("},%d,%d)\n",(hs)[i].len_fr,(hs)[i].to);
	}
}

/*   ## CPU ##
Read graph from file
*/
/*
Input:
	FILE : relative path and file name
	Vertices : pointer of pointer of integers 
	num_vertices : number of integers pointed from *Vertices
	Edges : pointer of pointer of Hyperarcs
	num_edges : number of hyperarcs pointed from *Edges
!!!!!!!
MODIFY:
	**Vertices, **Edges, num_edges, num_vertices
*/
void readGraph(std::string FILE, int** Vertices, int &num_vertices, Hyperarc ** Edges, int &num_edges, Hypervector** initial, int &num_initial){
	std::ifstream file_graph;
	std::string line, pref, from;
	int idxE = 0, idxV = 0, len_fr, to, temp, temp1,temp2;
	std::map<std::string, int> temporaneo;
	
	file_graph.open(FILE, std::ios::in);
	if (file_graph.is_open())
	{
		while ( std::getline(file_graph,line) )
		{
			pref = line.substr(0,3);
			if(pref=="INI"){
				num_vertices = std::stoi(line.substr(3, (int)line.find(",")-3));
				num_edges = std::stoi(line.substr(line.find(",")+1));
				
				printf("Vertici %d, Edges %d ", num_vertices, num_edges);
				
				*Vertices = (int*) malloc(sizeof(int)*num_vertices);
				*Edges = (Hyperarc*) malloc(sizeof(Hyperarc)*num_edges);
				
				printf("OK\n");
			}else if(pref == "(HA"){
				temp = line.find("}");
				temp2 = line.find("{");
				from = (line.substr(temp2+1, temp-(temp2+1)));
				temp1 = line.find(",",temp+1);
				temp2 = line.find(",", temp1+1);
				len_fr = std::stoi(line.substr(temp1+1, temp2-temp));
				
				to = std::stoi(line.substr(temp2+1, line.find(")")-(temp2+1)));
				(*Edges)[idxE] = {(int*)malloc(sizeof(int)*len_fr), NULL, len_fr, to};
				temp = 0;
				for(int i=0; i<len_fr; i++){
					temp1 = from.find(",", temp);
					if(temp==-1)
						(*Edges)[idxE].from[i] = std::stoi(from.substr(temp));
					else
						(*Edges)[idxE].from[i] = std::stoi(from.substr(temp,temp1));
					temp=temp1+1;
					
				}
				if(temporaneo.find(from)==temporaneo.end())
					temporaneo.insert(make_pair(from,len_fr));
				idxE++;
			}else if(pref == "(VE"){
				(*Vertices)[idxV] = std::stoi(line.substr(3, line.find(")")-1));
				idxV ++;
			}
		}
		*initial = (Hypervector*) malloc(sizeof(Hypervector*)*temporaneo.size());
		int itera=0;
		
		for(auto it=temporaneo.begin(); it!=temporaneo.end(); it++){
			temp=0; temp1=0;
			(*initial)[itera] = {(int*)malloc(sizeof(int)*it->second),it->second};
			for(int j=0; j<it->second; j++){
				temp1=it->first.find(",", temp);
				(*initial)[itera].vectors[j] =  std::stoi(it->first.substr(temp,temp1-temp));
				temp = temp1+1;
			}
			
			itera++;
		}
			
		num_initial = temporaneo.size();
		file_graph.close();
	}
	
}

/*   ## CPU ##
Write graph from file
*/
/*
Input:
	Vertices : pointer of integers 
	num_vertices : number of integers pointed from Vertices
	Edges : pointer of Hyperarcs
	num_edges : number of hyperarcs pointed from Edges
	FILE : relative path and file name
*/
void writeGraph(int* Vertices, int num_vertices, Hyperarc* Edges, int num_edges, std::string FILE){
	std::ofstream myFile;
	myFile.open(FILE);
	myFile << "INI " << num_vertices <<"," << num_edges <<"\n";
	for(int i=0; i<num_vertices; i++)
		myFile << "(VE " << i <<")\n";
	for(int i=0; i<(num_edges); i++){
		myFile << "(HA {";
		for(int j=0; j<(Edges)[i].len_fr; j++){
			myFile << (Edges)[i].from[j];
			if(j!=((Edges)[i].len_fr)-1) myFile << ",";
		}
		myFile << "}," << (Edges)[i].len_fr<<","<<(Edges)[i].to <<")\n";
			
	}
	
	
	
	myFile.close();
}	

/*   ## CPU ##
Write operation time to file
*/
/*
Input:
	num_vertices : number of integers pointed from Vertices
	num_edges : number of hyperarcs pointed from Edges
*/
void writeTime(int num_vertices, int num_edges){
	std::ofstream myFile;
	myFile.open("TimeSave.txt", std::fstream::app);
	myFile << "<G " << num_vertices <<"," << num_edges <<" ("<<durata<<" ns)>\n";
	myFile.close();
}


__device__ bool equalHyperVector(int * hyperarcs, int * vectors, int minLen){
	bool ok=true;
	for(int i=0; i<minLen && ok; i++){
		ok=hyperarcs[i]==vectors[i];
	}
	return ok;
}


/*   ## GPU ##
Find neighbors during one BFS step
*/
/*
Input:
	Vertices : pointer of integers 
	num_vertices : number of integers pointed from Vertices
	Edges : pointer of Hyperarcs
	num_edges : number of hyperarcs pointed from Edges
	FrontierUpdate : tracks changes during BFS step
	Visited : tracks the visited nodes during BFS
	Cost : the distance between start node and the other node
	thidLast : the node from to which to search for hyperarcs
!!!!!!
MODIFY:
	Cost, Visited
*/
__global__ void neighOp(int *Vertices, int num_vertices, Hyperarc * Edges, int num_edges, bool* FrontierUpdate, bool* Visited, int* Cost, Hypervector thidLast, int first){
	int thid = blockIdx.x * blockDim.x + threadIdx.x;
	bool thereIs = false;
	int thidI;
	
	
	for(int Pass=0; Pass<ceilf((num_edges/(blockDim.x)))+2; Pass++){
		thidI = thid + Pass*blockDim.x;
	
		if(thidI<num_edges){
			for(int i=0; i<num_vertices; i++){
				thereIs = false;	
				if(Edges[thidI].to==Vertices[i]){
										
						if(equalHyperVector(Edges[thidI].from_dev,thidLast.vectors,thidLast.len) && first==1){
							if(Visited[Vertices[i]]==false){
								Cost[Vertices[i]] = Cost[thidLast.vectors[0]]+1;
								FrontierUpdate[Vertices[i]] = true;
							}
						}else if(first!=1){
							for(int j=0; j<Edges[thidI].len_fr && !thereIs; j++){
								thereIs = Edges[thidI].from_dev[j]==thidLast.vectors[0];
							}
							if(thereIs){
								if(Visited[Vertices[i]]==false){
									Cost[Vertices[i]] = Cost[thidLast.vectors[0]]+1;
									FrontierUpdate[Vertices[i]] = true;
								}
							}
						}
					
				}
			}
		}
	}
	__syncthreads();

}


/*   ## GPU ##
One step of BFS 
*/
/*
Input:
	Vertices : pointer of integers 
	num_vertices : number of integers pointed from Vertices
	Edges : pointer of Hyperarcs
	num_edges : number of hyperarcs pointed from Edges
	Frontier : the Frontier of the step
	FrontierUpdate : tracks changes during BFS step
	Visited : tracks the visited nodes during BFS
	Cost : the distance between start node and the other node
!!!!!!
MODIFY:
	Frontier, Cost, Visited
!!!!!!
Kernel invoked (1 GPU block)
*/	
__global__ void bfs(int *Vertices, int num_vertices, Hyperarc * Edges,const int num_edges, bool * Frontier, bool* FrontierUpdate, bool* Visited, int* Cost, int* first, int* len){
	int thid = blockIdx.x * blockDim.x + threadIdx.x;
	int thidI, idx=0;
	
	if(*len!=-1){
		if(thid==0)
			neighOp<<< 1,min(num_edges,MAX_THREADS)>>>(Vertices, num_vertices,  Edges, num_edges, FrontierUpdate, Visited, Cost, {first,*len}, 1);
	}else
		for(int Pass=0; Pass<ceilf(num_vertices/(blockDim.x))+2; Pass++){
			thidI = thid + Pass*(blockDim.x);
			if(thidI < num_vertices && Frontier[thidI]){
				Frontier[thidI] = false;
				neighOp<<< 1,min(num_edges,MAX_THREADS)>>>(Vertices, num_vertices,  Edges, num_edges, FrontierUpdate, Visited, Cost, {new int[]{thid},1}, 0);
			}
			
		}
	
	__syncthreads();
	
}


/*   ## GPU ##
Update the ausiliary structur of BFS
*/
/*
Input:
	num_vertices : number of integers pointed from Vertices
	FrontierUpdate : tracks changes during BFS step
	Visited : tracks the visited nodes during BFS
	Cost : the distance between start node and the other node
	next : check wheater to proceed or not
!!!!!!
MODIFY:
	Frontier, Visited, next, FrontierUpdate
*/
__global__ void bfs_update(int num_vertices, bool * Frontier, bool * FrontierUpdate, bool * Visited, int * next){
	int thid = blockIdx.x * blockDim.x + threadIdx.x;
	int thidI;
	for(int Pass=0; Pass<ceilf((num_vertices/(blockDim.x)))+2; Pass++){
		thidI = thid + Pass*(blockDim.x );
		if(thidI<num_vertices && FrontierUpdate[thidI]){
			Frontier[thidI] = true;
			Visited[thidI] = true;
			*next = 1;
			FrontierUpdate[thidI] = false;
			
		}
	}
	__syncthreads();
	
}


/*   ## CPU ##
Calculate the neighbors through BFS on GPU
*/
/*
Input:
	Vertices : pointer of integers 
	num_vertices : number of integers pointed from Vertices
	Edges : pointer of Hyperarcs
	num_edges : number of hyperarcs pointed from Edges
	node : the starting node of BFS
Return:
	Succint Closure hyperarcs
!!!!!!
MODIFY:
	Edges
!!!!!!
Sequentially
	Kernel invoked (MAX_BLOCKS GPU block)
	Kernel invoked (MAX_BLOCKS GPU block)
!!!!!!
Allocate num_vertices*3 bool array CPU/GPU
Allocate num_vertices*1 int  array CPU
Allocate num_vertices*2 int  array GPU
Allocate num_edges Hyperarc  array GPU
Allocate new Edges array that contains the succint closure hyperarcs
Allocate Temporary Array to copy to GPU the set of node of the hyperarcs

*/
Hyperarc * Edges_DEV;
int * Vertices_DEV;

Hyperarc * graph_bfs_nieces(int * Vertices, int num_vertices, Hyperarc * Edges, const int num_edges, Hypervector node){
	int * Cost_HOS, *Cost_DEV;
	bool * Frontier_HOS, *Frontier_DEV;
	bool * FrontierUpdate_HOS, *FrontierUpdate_DEV;
	bool * Visited_HOS, *Visited_DEV;
	
	int * from;
	Hyperarc *newEdges;
	int sizeEdges=0;
	
	Cost_HOS = (int*) malloc(sizeof(int)*num_vertices);
	gpuErrchk(hipMalloc((void**)&Cost_DEV, sizeof(int)*num_vertices));
	
	Frontier_HOS = (bool*) malloc(sizeof(bool)*num_vertices);
	gpuErrchk(hipMalloc((void**)&Frontier_DEV, sizeof(bool)*num_vertices));
	
	FrontierUpdate_HOS = (bool*) malloc(sizeof(bool)*num_vertices);
	gpuErrchk(hipMalloc((void**)&FrontierUpdate_DEV, sizeof(bool)*num_vertices));
	
	Visited_HOS = (bool*) malloc(sizeof(bool)*num_vertices);
	gpuErrchk(hipMalloc((void**)&Visited_DEV, sizeof(bool)*num_vertices));
	
	printf("ORA 0\n");
	
	for(int i=0; i<num_vertices; i++){
		Cost_HOS[i] = -1;
		Frontier_HOS[i] = false;
		FrontierUpdate_HOS[i] = false;
		Visited_HOS[i] = false;
	}
	for(int i=0; i<node.len; i++){
		Frontier_HOS[node.vectors[i]] 	= true;
		Visited_HOS[node.vectors[i]] 	= true;
		Cost_HOS[node.vectors[i]] 		= 0;
	}
	
	hipMemcpy(Cost_DEV, Cost_HOS, sizeof(int)*num_vertices, hipMemcpyHostToDevice);
	hipMemcpy(Frontier_DEV, Frontier_HOS, sizeof(bool)*num_vertices, hipMemcpyHostToDevice);
	hipMemcpy(FrontierUpdate_DEV, FrontierUpdate_HOS, sizeof(bool)*num_vertices, hipMemcpyHostToDevice);
	hipMemcpy(Visited_DEV, Visited_HOS, sizeof(bool)*num_vertices, hipMemcpyHostToDevice);
	
	
	int *next_HOS, *next_DEV, *len_HOS, *len_DEV;
	
	gpuErrchk(hipMalloc((void**) &next_DEV, sizeof(int)));
	gpuErrchk(hipMalloc((void**) &len_DEV, sizeof(int)));
	
	
	
	
	next_HOS = (int*) malloc(sizeof(int));
	len_HOS = (int*) malloc(sizeof(int));
	
	gpuErrchk(hipMalloc((void**)&from, sizeof(int)*node.len));
			
	gpuErrchk(hipMemcpy(from, node.vectors,  sizeof(int)*node.len, hipMemcpyHostToDevice));
	
	*len_HOS = node.len;
	gpuErrchk(hipMemcpy(len_DEV, len_HOS,  sizeof(int), hipMemcpyHostToDevice));
	

	
	*next_HOS = 1;
	
	
	while(*next_HOS==1){
		*next_HOS = 0;
		
		gpuErrchk(hipMemcpy(next_DEV, next_HOS, sizeof(int), hipMemcpyHostToDevice));
		
		bfs<<<MAX_BLOCKS, min(num_vertices, MAX_THREADS) >>>(Vertices_DEV, num_vertices, Edges_DEV, num_edges, Frontier_DEV, FrontierUpdate_DEV, Visited_DEV, Cost_DEV, from, len_DEV);
		
		gpuErrchk( hipPeekAtLastError() );
		hipDeviceSynchronize();
		
		bfs_update<<<MAX_BLOCKS, min(num_vertices, MAX_THREADS) >>>(num_vertices, Frontier_DEV, FrontierUpdate_DEV, Visited_DEV, next_DEV);
		gpuErrchk( hipPeekAtLastError() );
		hipDeviceSynchronize();
		gpuErrchk(hipMemcpy(next_HOS, next_DEV , sizeof(int), hipMemcpyDeviceToHost));
		
		*len_HOS = -1;
		
		gpuErrchk(hipMemcpy(len_DEV, len_HOS, sizeof(int), hipMemcpyHostToDevice));
		

		
	}
	
	
	gpuErrchk(hipFree(from));
	gpuErrchk(hipFree(next_DEV));
	gpuErrchk(hipFree(len_DEV));
	gpuErrchk(hipFree(Frontier_DEV));
	gpuErrchk(hipFree(FrontierUpdate_DEV));
	gpuErrchk(hipFree(Visited_DEV));
	
	(free(Frontier_HOS));
	(free(FrontierUpdate_HOS));
	(free(Visited_HOS));
	
	hipMemcpy(Cost_HOS, Cost_DEV, sizeof(int)*num_vertices, hipMemcpyDeviceToHost);
	gpuErrchk(hipFree(Cost_DEV));
	
	for(int i=0; i<num_vertices; i++){
		if(Cost_HOS[Vertices[i]]>1) sizeEdges++;
	}
	
	
	
	newEdges = (Hyperarc*) malloc(sizeof(Hyperarc)*(sizeEdges+1));
	int k=0;
	for(int i=0; i<num_vertices && k<sizeEdges; i++){
		if(Cost_HOS[Vertices[i]]>1){
			newEdges[k] = {(int*)malloc(sizeof(int)*node.len),NULL,node.len,Vertices[i]};
			std::copy(node.vectors,(node.vectors)+(node.len), newEdges[k].from);
						
			gpuErrchk(hipMalloc((void**)&from, sizeof(int)*node.len));
			gpuErrchk( hipPeekAtLastError() );
		
			hipMemcpy(from, node.vectors, sizeof(int), hipMemcpyHostToDevice);
			gpuErrchk( hipPeekAtLastError() );
		
			newEdges[k].from_dev = from;
						
			k++;
		}
	}
	
	
	newEdges[k] = {NULL,NULL,-1,-1};
	
	return newEdges;
}



/*   ## CPU ##
Force the copy of array inside another (struct of Hyperarcs)
*/
/*
Input:
	a : pointer of Hyperarcs (to)
	b : pointer of Hyperarcs (from)
	w : size of a and b 
return:
	if the copy ended 0, anything else
!!!!!!
MODIFY:
	a
!!!!!!
MULTI PARALLELISM OPENMP
*/
int copyaa(Hyperarc* a, Hyperarc* b, int w){
	int i;
	/*
	#pragma omp parallel for private(i) shared(a,b) num_threads(nThr)
	for(i=0; i<w; i++){
		a[i] = b[i];
	}
	*/
	//std::memcpy(a,b,w*sizeof(Hyperarc));
	
	//using std c++17
	std::copy(std::execution::par,b,b+w,a);
	
	std::sort(a, a + w, compareTwoHyerarch);
	std::unique( a, a + w , ne_compareTwoHyerarch);
	return 0;
}



/*   ## CPU ##
Prepare the data and lunch BFS
*/
/*
Input:
	Vertices : pointer of integers 
	num_vertices : number of integers pointed from Vertices
	Edges : pointer of pointer of Hyperarcs
	num_edges : number of hyperarcs pointed from Edges
	Start : pointer of Integers (the vertices from which to start the BFS)
	num_start : number of integers pointed from Start
!!!!!!
MODIFY:
	Edges
!!!!!!
MULTI PARALLELISM OPENMP
!!!!!!
Allocate (num_start)* int array CPU (split between threads)

*/
void gpu_bfs(int * Vertices, int num_vertices, Hyperarc ** Edges, int &num_edges, Hypervector * Start, int num_start){
	Hyperarc ** nArch, * newEdges;
	int nNew_Arch=0, *sizeTot;
	int * from;
	
	

	gpuErrchk(hipMalloc((void**)&Vertices_DEV, sizeof(int)*num_vertices));
	gpuErrchk(hipMalloc((void**)&Edges_DEV, sizeof(Hyperarc)*num_edges));
	
	hipMemcpy(Vertices_DEV, Vertices, sizeof(int)*num_vertices, hipMemcpyHostToDevice);
	int len_frT = 0;
	for(int i=0; i<num_edges; i++){	
		len_frT = (*Edges)[i].len_fr;
		gpuErrchk(hipMalloc((void**)&from, sizeof(int) * len_frT));
		
		hipMemcpy(from, (*Edges)[i].from, sizeof(int)* len_frT, hipMemcpyHostToDevice);
		
		
		(*Edges)[i].from_dev = from;
		
	}
	
	hipMemcpy(Edges_DEV, *Edges, sizeof(Hyperarc)*num_edges, hipMemcpyHostToDevice);
	
	
	
	#pragma omp parallel num_threads(nThr) shared(Vertices, Edges, Start, num_vertices, num_edges, num_start, nNew_Arch, sizeTot, newEdges) private(nArch) 
	{
		if(omp_get_thread_num()==0) sizeTot = (int*) malloc(sizeof(int)*omp_get_num_threads());
		
		int workN = ceil((double)num_start/(double)(omp_get_num_threads()));
		int ini = workN * omp_get_thread_num();
		int end = fmin(workN * omp_get_thread_num() + workN, num_vertices);
		int * mySyze = (int*)malloc(sizeof(int)*(end-ini));
		int mySSyze=0;
		
	
		
		#ifdef DEBUG
		printf("%d (%d,%d)\n",omp_get_thread_num(),ini,end);
		#endif
		
		nArch = (Hyperarc**)malloc(sizeof(Hyperarc*)*workN);
		
		for(int i=ini; i<end; i++){
			mySyze[i-ini]=0;
			nArch[i-ini] = graph_bfs_nieces(Vertices, num_vertices, *Edges, num_edges, Start[i]);
			
			for(int j=0; j<num_vertices && (nArch[i-ini][j].to!=-1); j++){
				#pragma omp atomic
				nNew_Arch+=1;
				
				mySSyze++;
				mySyze[i-ini]++;
			}
		}
		#ifdef DEBUG
		printf("%d ",omp_get_thread_num());
		#endif
		
		sizeTot[omp_get_thread_num()] = mySSyze;
		
		#pragma omp barrier
		if(omp_get_thread_num()==0){
			newEdges = (Hyperarc*) malloc(sizeof(Hyperarc)*(nNew_Arch+num_edges));
			#ifdef DEBUG
			printf("\n");
			#endif
		}
		
		#pragma omp barrier
		int idx=0,ida,len=end-ini;
		
		ini=0;
		for(int i=0; i<omp_get_thread_num(); i++) ini+=sizeTot[i];
		
		#ifdef DEBUG
		printf("%d: ini %d-%d (%d)\n", omp_get_thread_num(), ini, ini+mySSyze, (nNew_Arch+num_edges));
		#endif
		
		ida=ini; 
		
		while(ida<ini+mySSyze){
			for(int j=0;j<len; j++){
				#pragma omp critical
				{
					if(idx<mySyze[j]){
						newEdges[ida] = nArch[j][idx];
						ida++;
						
					
					}
				}
				
			}
			idx++;
			
		}
		#ifdef DEBUG
		printf("finito %d\n",omp_get_thread_num());
		#endif
		#pragma omp barrier
	}
	
	gpuErrchk(hipFree(Edges_DEV));
	gpuErrchk(hipFree(Vertices_DEV));
	
	#ifdef DEBUG
	printf("copia finale\n");
	#endif
	
	for(int i=0; i<num_edges; i++)
		newEdges[i+nNew_Arch] = (*Edges)[i];
	
	num_edges += nNew_Arch;
	free(*Edges);
	
	
	
	printf("inizio copia");
	(*Edges) = (Hyperarc*) malloc(sizeof(Hyperarc*)*num_edges);
	(*Edges) = newEdges;
	printf(" %s",*Edges==NULL? "error":"");
	printf("%s\n",copyaa((*Edges),(newEdges), (num_edges))==0?"terminata":" error");
	
	
	
}


/*   ## CPU ##
main function
read graph and initialize it, lunch gpu_bfs, write graphs
(writing on console or file depends on the type of compilation)
*/
/*
Input:
	args = { name_program, [name_graphs] }

*/
int main(int argn, char ** args){
	
	
	
	
	system("cls");
	
	int ** Vertices /*= (int*) malloc(sizeof(int)*8);*/ = (int**) malloc(sizeof(int*));
	int num_vertices=0, num_edges=0, num_initial=0;
	bool argg=false;
	std::string file_name;
	Hyperarc ** Edges  = (Hyperarc**) malloc(sizeof(Hyperarc*));
	Hypervector ** initial = (Hypervector**) malloc(sizeof(Hypervector*));
	
	if(argn>1){
		for(int i=0; i<argn; i++)
			if(std::string(args[i]).substr(0,4) == "-nT="){
				nThr = std::stoi(std::string(args[i]).substr(4));
				argg=true;
			}
			else 
				file_name = args[i];
		if(!argg){
			#ifndef NTHR
				#pragma omp parallel shared(nThr)
				{
					if(omp_get_thread_num()==0)
						nThr = omp_get_max_threads();
				}
			#endif
		}
		#ifdef TIME
		begin = std::chrono::high_resolution_clock::now();
		#endif
		readGraph(file_name,Vertices,num_vertices, Edges, num_edges, initial, num_initial);
		
		#ifdef TIME
		end = std::chrono::high_resolution_clock::now();
		#endif
			
	}else{
		
		#ifdef TIME
		begin = std::chrono::high_resolution_clock::now();
		#endif
		readGraph("prova.txt",Vertices,num_vertices, Edges, num_edges, initial, num_initial);
		
		#ifdef TIME
		end = std::chrono::high_resolution_clock::now();
		#endif
	}
	
	
	
	#ifdef TIME
	durataRead = std::chrono::duration_cast<std::chrono::nanoseconds>( end - begin ).count();
	#ifdef DEBUG
	printf("reading time: %lu ns\n",durataRead);
	printf("CPU threads: %d\nGPU Block:%d\nGPU threads: %d\n", nThr,MAX_BLOCKS,MAX_THREADS);
	#endif
	#endif
	
	#ifndef HIDE
	for(int i=0; i<(num_edges); i++){
		printf("(HA {");
		for(int j=0; j<(*Edges)[i].len_fr; j++){
			printf("%d",(*Edges)[i].from[j]);
			if(j!=((*Edges)[i].len_fr)-1) printf(",");
		}
		printf("} %d)\n",(*Edges)[i].to);
			
	}
	
	for(int i=0; i<num_vertices; i++)
		printf("(VE %d)\n",(*Vertices)[i]);
	
	for(int i=0; i<(num_initial); i++){
		printf("(HV {");
		for(int j=0; j<(*initial)[i].len; j++){
			printf("%d",(*initial)[i].vectors[j]);
			if(j!=((*initial)[i].len)-1) printf(",");
		}
		printf("})\n");
			
	}
	#endif
	
	#ifndef NO_INPUT
	printf("Press ENTER to start\n");
	
	getchar();
	#endif
	#ifdef TIME
	begin = std::chrono::high_resolution_clock::now();
	#endif
	gpu_bfs(*Vertices, num_vertices, Edges, num_edges, *initial, num_initial);

	#ifdef TIME
	end = std::chrono::high_resolution_clock::now();
	durata = std::chrono::duration_cast<std::chrono::nanoseconds>( end - begin).count();
	#endif
	
	
	
	#ifdef DEBUG
	printf("END\n");
	#ifdef TIME 
	printf("::durata: %lu ns\n",durata);
	#endif
	#endif
	#ifndef HIDE
	
	for(int i=0; i<(num_edges); i++){
		printf("(HA {");
		for(int j=0; j<(*Edges)[i].len_fr; j++){
			printf("%d",(*Edges)[i].from[j]);
			if(j!=((*Edges)[i].len_fr)-1) printf(",");
		}
		printf("},%d,%d)\n",(*Edges)[i].len_fr,(*Edges)[i].to);
	}
	#endif
	
	#ifdef FILE_OUT
	if(argn>1){
		
		file_name = (file_name.substr(0,file_name.find(".", 2)) + ("_new.txt"));
		writeGraph(*Vertices,num_vertices, *Edges, num_edges, file_name );
	}else
		writeGraph(*Vertices,num_vertices, *Edges, num_edges,"prova_new.txt");
	#ifdef TIME
		writeTime(num_vertices,num_edges);
	#endif
	#endif
	
}
	
	
